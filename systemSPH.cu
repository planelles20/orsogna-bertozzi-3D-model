#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------
// Code file of SistemSPH class which simulate Vicsek model using
// Smoothed-particle hydrodynamics (SPH) method.
//
// Licensing: This code is distributed under the Apache License 2.0
// Author: Carlos Planelles Alemany, planelles20(at)gmail(dot)com
//-----------------------------------------------------------------------

#include "systemSPH.h"

////////////////////////////////  kernels  ////////////////////////////////////

__global__ void clearGridIndices_kernel(intvec2 *d_inidices, int numCells) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numCells){
        d_inidices[i].init = 0;
        d_inidices[i].end  = 0;
    }
}

__global__ void builtGridIncices_kernel(intvec2 *gridIdx, particle *d_particle, int numParticles){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numParticles){
        if( i==0 && d_particle[i].cellIdx != d_particle[i+1].cellIdx){
            gridIdx[d_particle[i].cellIdx].init = 0;
        }

        if(i!=0 && d_particle[i].cellIdx != d_particle[i-1].cellIdx){
            gridIdx[d_particle[i].cellIdx].init = i;
        }
        if(i!=(numParticles-1) && d_particle[i].cellIdx != d_particle[i+1].cellIdx){
            gridIdx[d_particle[i].cellIdx].end = i;
        }
        if(i==(numParticles-1) && d_particle[i].cellIdx == d_particle[i-1].cellIdx){
            gridIdx[d_particle[i].cellIdx].end = i;
        }
    }
}

__global__ void posParticleCell_kernel(particle *d_particle, int numParticles, int xMesh, int yMesh, int zMesh){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<numParticles){
        int ii = int(d_particle[i].x*float(xMesh));
        int jj = int(d_particle[i].y*float(yMesh));
        int kk = int(d_particle[i].z*float(zMesh));
        if(ii == xMesh) ii--;
        if(jj == yMesh) jj--;
        if(kk == zMesh) kk--;
        d_particle[i].cellIdx = ii+jj*xMesh+kk*xMesh*yMesh;
    }
}

__global__ void bitonic_sort_kernel(particle *dev_values, int j, int k) {
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i].cellIdx>dev_values[ixj].cellIdx) {
        /* exchange(i,ixj) particles; */
        //cell
        particle temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i].cellIdx<dev_values[ixj].cellIdx) {
        /* exchange(i,ixj); */
        particle temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

__device__ float DU(float xi, float xj){
    float d = abs(xi-xj)/0.001;
    float Cr = 0.6;
    float Ca = 1.0;
    float lr = 0.5;
    float la = 21.0;
    return 1000.0*(-Cr/lr*exp(-d/lr)+Ca/la*exp(-d/la));
    //return +1000.0;
}

__global__ void calcul_kernel(particle *d_particle, intvec2 *d_inidices,
                              calcStruct *d_calc, int numParticles,
                              int xMesh, int yMesh, int zMesh){

    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numParticles){
        //
        int N = 0;
        float DUmedia_x = 0.0,  DUmedia_y = 0.0,  DUmedia_z = 0.0;
        float dR = 1.0/float(xMesh);
        float dnorm_x, dnorm_y, dnorm_z, norm;
        // look own cell
        for(int ii=d_inidices[d_particle[i].cellIdx].init; ii<d_inidices[d_particle[i].cellIdx].end; ++ii){
            if((abs(d_particle[ii].x-d_particle[i].x)<dR)
            && (abs(d_particle[ii].y-d_particle[i].y)<dR)
            && (abs(d_particle[ii].z-d_particle[i].z)<dR)
            && i!=ii){
               ++N;
               norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                           pow(d_particle[ii].y-d_particle[i].y, 2)+
                           pow(d_particle[ii].z-d_particle[i].z, 2));
               dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
               dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
               dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
               DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
               DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
               DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
            }
        }
        /// looking near cells
        // x+1
        if(d_particle[i].cellIdx+1 < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1].init; ii<d_inidices[d_particle[i].cellIdx+1].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1
        if(d_particle[i].cellIdx-1 > 0){
            for(int ii=d_inidices[d_particle[i].cellIdx-1].init; ii<d_inidices[d_particle[i].cellIdx-1].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // y+1
        if(d_particle[i].cellIdx+1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // y-1
        if(d_particle[i].cellIdx-1*xMesh > 0){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // z+1
        if(d_particle[i].cellIdx+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // z-1
        if(d_particle[i].cellIdx-1*xMesh*yMesh > 0){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y+1
        if(d_particle[i].cellIdx+1+1*xMesh > 0 && d_particle[i].cellIdx+1+1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y-1
        if(d_particle[i].cellIdx+1-1*xMesh > 0 && d_particle[i].cellIdx+1-1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y+1
        if(d_particle[i].cellIdx-1+1*xMesh > 0 && d_particle[i].cellIdx-1+1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y-1
        if(d_particle[i].cellIdx-1-1*xMesh > 0 && d_particle[i].cellIdx-1-1*xMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y+1, z+1
        if(d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y-1, z+1
        if(d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y+1, z+1
        if(d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y-1, z+1
        if(d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y+1, z-1
        if(d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y-1, z-1
        if(d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y+1, z-1
        if(d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y-1, z-1
        if(d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y, z+1
        if(d_particle[i].cellIdx+1+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y, z+1
        if(d_particle[i].cellIdx-1+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x, y+1, z+1
        if(d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x, y-1, z+1
        if(d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh+1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x+1, y, z-1
        if(d_particle[i].cellIdx+1-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x-1, y, z-1
        if(d_particle[i].cellIdx-1-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x, y+1, z-1
        if(d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx+1*xMesh-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }
        // x, y-1, z-1
        if(d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh > 0 && d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh < xMesh*yMesh*zMesh){
            for(int ii=d_inidices[d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh].init; ii<d_inidices[d_particle[i].cellIdx-1*xMesh-1*xMesh*yMesh].end; ++ii){
                if((abs(d_particle[ii].x-d_particle[i].x)<dR)
                && (abs(d_particle[ii].y-d_particle[i].y)<dR)
                && (abs(d_particle[ii].z-d_particle[i].z)<dR)){
                    ++N;
                    norm = sqrt(pow(d_particle[ii].x-d_particle[i].x, 2)+
                                pow(d_particle[ii].y-d_particle[i].y, 2)+
                                pow(d_particle[ii].z-d_particle[i].z, 2));
                    dnorm_x = (d_particle[i].x-d_particle[ii].x)/norm;
                    dnorm_y = (d_particle[i].y-d_particle[ii].y)/norm;
                    dnorm_z = (d_particle[i].z-d_particle[ii].z)/norm;
                    DUmedia_x += DU(d_particle[i].x, d_particle[ii].x)*dnorm_x;
                    DUmedia_y += DU(d_particle[i].y, d_particle[ii].y)*dnorm_y;
                    DUmedia_z += DU(d_particle[i].z, d_particle[ii].z)*dnorm_z;
                }
            }
        }

        // calc
        if(N != 0){
            d_calc[i].DU.x = DUmedia_x/N;
            d_calc[i].DU.y = DUmedia_y/N;
            d_calc[i].DU.z = DUmedia_z/N;
            d_calc[i].N = N;
        }
        else {
            d_calc[i].DU.x = 0.0;
            d_calc[i].DU.y = 0.0;
            d_calc[i].DU.z = 0.0;
            d_calc[i].N = 0;
        }

    }
}

__global__ void integrate_kernel(particle *d_particle, calcStruct *d_calc, int numParticles, float seed){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < numParticles){

        hiprandState_t state;
        hiprand_init(seed,
                    i,
                    0,
                    &state);

        float dt =0.001;
        float alpha = 1.0;
        float beta = 0.5*30.0;
        float mod_v = sqrt(pow(d_particle[i].vx,2)+
                           pow(d_particle[i].vy,2)+
                           pow(d_particle[i].vz,2));
        // dv
        //d_particle[i].theta = d_calc[i].thetaMedium + 0.1*hiprand_normal(&state);
        //d_particle[i].alpha = d_calc[i].alphaMedium + 0.1*hiprand_normal(&state);

        float dvx = (alpha-beta*pow(mod_v,2))*d_particle[i].vx-d_calc[i].DU.x;
        float dvy = (alpha-beta*pow(mod_v,2))*d_particle[i].vy-d_calc[i].DU.y;
        float dvz = (alpha-beta*pow(mod_v,2))*d_particle[i].vz-d_calc[i].DU.z;

        // v
        float vx = d_particle[i].vx + dvx*dt;
        float vy = d_particle[i].vy + dvy*dt;
        float vz = d_particle[i].vz + dvz*dt;

        d_particle[i].vx = vx;
        d_particle[i].vy = vy;
        d_particle[i].vz = vz;

        // x
        if(d_particle[i].x<0.0 && vx<0.0) {
            d_particle[i].x = 1.0;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].x>1.0 && vx > 0.0) {
            //theta = pi-theta;
            d_particle[i].x = 0.0;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].y<0.0 && vy < 0.0) {
            //theta = 2.0*pi-theta;
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = 1.0;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].y>1.0 && vy > 0.0) {
            //theta = 2.0*pi-theta;
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = 0.0;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
        else if(d_particle[i].z<0.0 && vz < 0.0) {
            //theta = 2.0*pi-theta;
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = 1.0;
        }
        else if(d_particle[i].z>1.0 && vz > 0.0) {
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = 0.0;
        }
        else {
            d_particle[i].x = d_particle[i].x+vx*dt;
            d_particle[i].y = d_particle[i].y+vy*dt;
            d_particle[i].z = d_particle[i].z+vz*dt;
        }
    }
}

/////////////////////////////  methods ////////////////////////////////////////

SystemSPH::SystemSPH(unsigned int blocks,
                     unsigned int threads,
                     unsigned int xMesh,
                     unsigned int yMesh,
                     unsigned int zMesh) {

    //glewInit();
    //glewExperimental = GL_TRUE;
    this->numBlocks = blocks;
    this->numThreads = threads;
    this->numParticles = blocks*threads;
    this->xMeshDim = xMesh;
    this->yMeshDim = yMesh;
    this->zMeshDim = zMesh;
    this->numIndices = xMesh*yMesh*zMesh;

    this->h_particle = new particle[this->numParticles];
    this->h_inidices = new intvec2[this->numIndices];
    this->h_calc = new calcStruct[this->numParticles];

    this->particleIndices = new GLushort[this->numParticles];

    //init particle values
    this->InitParticleData();

    //Generate openGL buffers
    //vertex array object
    //Create vertex buffer object(s)+
    glGenVertexArrays(1, &this->VAO);
    //Bind the Vertex Array Object first, then bind and set vertex buffer(s) and attribute pointer(s).
    glBindVertexArray(this->VAO);


    // set vertex buffer
    glGenBuffers(1, &this->VBOparticles);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOparticles);
    glBufferData(GL_ARRAY_BUFFER, this->numParticles*sizeof(particle), this->h_particle, GL_DYNAMIC_COPY); //like cupdamemcy host->device
    hipGraphicsGLRegisterBuffer(&this->cudaResourceBufParticles, this->VBOparticles, hipGraphicsRegisterFlagsNone);

    glGenBuffers(1, &this->EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, this->numParticles*sizeof(GLushort), this->particleIndices, GL_STATIC_DRAW);

    // bind attribute pointer(s)
    // Position attribute
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(particle), (GLvoid*)(2 * sizeof(GLint)));
    glEnableVertexAttribArray(0);
    // Velocity attribute
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(particle), (GLvoid*)(2*sizeof(GLint)+3*sizeof(GLfloat)));
    glEnableVertexAttribArray(1);

    //glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0); //unbind EBO
    glBindBuffer(GL_ARRAY_BUFFER, 0); // unbind VBO
    glBindVertexArray(0); // Unbind VAO

    glGenBuffers(1, &this->VBOindices);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOindices);
    glBufferData(GL_ARRAY_BUFFER, this->numIndices*sizeof(intvec2), this->h_inidices, GL_DYNAMIC_COPY); //like cupdamemcy host->device
    hipGraphicsGLRegisterBuffer(&this->cudaResourceBufIndices, this->VBOindices, hipGraphicsRegisterFlagsNone);


    glGenBuffers(1, &this->VBOcalc);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOcalc);
    glBufferData(GL_ARRAY_BUFFER, this->numParticles*sizeof(calcStruct), this->h_calc, GL_DYNAMIC_COPY); //like cupdamemcy host->device
    hipGraphicsGLRegisterBuffer(&this->cudaResourceBufCalc, this->VBOcalc, hipGraphicsRegisterFlagsNone);

    /// boundaries
    // allocate memory
    this->boundaryPoints = new point[8];
    this->boundaryIndices = new GLushort[16]; // points per squares dot num of squares
    //create boundaries (init)
    this->CreateBounderiesPoints();
    this->CreateBounderiesIndices();

    //vertex array object
    glGenVertexArrays(1, &this->VAOboundary);
    glBindVertexArray(this->VAOboundary);


    //Create vertex buffer object
    glGenBuffers(1, &this->VBOboundary);
    glBindBuffer(GL_ARRAY_BUFFER, this->VBOboundary);
    glBufferData(GL_ARRAY_BUFFER, sizeof(*this->boundaryPoints)*8, this->boundaryPoints, GL_STATIC_DRAW);

    //Create Element Buffer Objects
    glGenBuffers(1, &this->EBOboundary);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->EBOboundary);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(*this->boundaryIndices)*16, this->boundaryIndices, GL_STATIC_DRAW);

    // Position attribute (3D)
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(*this->boundaryPoints), (GLvoid*)0);
    glEnableVertexAttribArray(0);

    glBindBuffer(GL_ARRAY_BUFFER, 0); // unbind VBOp
    glBindVertexArray(0); // Unbind VAOp
}

SystemSPH::~SystemSPH() {
    //hipFree(d_particle);
    //hipFree(d_inidices);
    glDeleteVertexArrays(1, &this->VAO);
    glDeleteBuffers(1, &this->VBOparticles);
    glDeleteBuffers(1, &this->VBOindices);
    glDeleteBuffers(1, &this->VBOcalc);
    glDeleteBuffers(1, &this->EBO);

    hipGraphicsUnregisterResource(this->cudaResourceBufParticles);
    hipGraphicsUnregisterResource(this->cudaResourceBufIndices);
    hipGraphicsUnregisterResource(this->cudaResourceBufCalc);
}

void SystemSPH::Particle_print() {
    //copy to host
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t size = this->numParticles*sizeof(particle);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &size, this->cudaResourceBufParticles);
    std::cout << hipMemcpy(this->h_particle, this->d_particle, this->numParticles*sizeof(particle), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);

    for (int i = 0; i < this->numParticles; ++i) {
        printf("Particle id: %d, particle cell id: %d, position: (%1.3f, %1.3f, %1.3f), velocity:  (%1.3f, %1.3f, %1.3f),",
                this->h_particle[i].id, this->h_particle[i].cellIdx,
                this->h_particle[i].x, this->h_particle[i].y, this->h_particle[i].z,
                this->h_particle[i].vx, this->h_particle[i].vy, this->h_particle[i].vz);
        printf("\n");
    }
}

void SystemSPH::Indices_print() {
    //copy to host
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t size = this->numIndices*sizeof(intvec2);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &size, this->cudaResourceBufIndices);
    hipMemcpy(this->h_inidices, this->d_inidices, this->numIndices*sizeof(intvec2), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);
    //print indices
    for (int i = 0; i < this->numIndices; ++i) {
      printf("Indice pos: %d, init: %d, end: %d, number of particles: %d",
              i, this->h_inidices[i].init, this->h_inidices[i].end,
              this->h_inidices[i].end-this->h_inidices[i].init);
      printf("\n");
    }
}

void SystemSPH::Calc_print() {
    //copy to host
    hipGraphicsMapResources(1, &this->cudaResourceBufCalc, 0);
    size_t size = this->numParticles*sizeof(calcStruct);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_calc, &size, this->cudaResourceBufCalc);
    std::cout << hipMemcpy(this->h_calc, this->d_calc, this->numParticles*sizeof(calcStruct), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufCalc, 0);
    //print indices
    for (int i = 0; i < this->numParticles; ++i) {
      printf("Particle index: %d. DU_x: %1.3f. DU_y: %1.3f. DU_z: %1.3f. # of neighbors: %d",
              i, this->h_calc[i].DU.x, this->h_calc[i].DU.y, this->h_calc[i].DU.z, this->h_calc[i].N);
      printf("\n");
    }
}

void SystemSPH::InitParticleData(){
    srand(time(NULL));
    for (int i = 0; i < this->numParticles; ++i) {
        this->particleIndices[i] = i;
        //particles
        this->h_particle[i].id = i;
        this->h_particle[i].x = 0.4*(float)rand()/(float)RAND_MAX+0.3;
        this->h_particle[i].y = 0.4*(float)rand()/(float)RAND_MAX+0.3;
        this->h_particle[i].z = 0.4*(float)rand()/(float)RAND_MAX+0.3;
        this->h_particle[i].vx = 0.01*(2.0*(float)rand()/(float)RAND_MAX-1.0);
        this->h_particle[i].vy = 0.01*(2.0*(float)rand()/(float)RAND_MAX-1.0);
        this->h_particle[i].vz = 0.01*(2.0*(float)rand()/(float)RAND_MAX-1.0);
    }
}

void SystemSPH::SortParticles(){

    //get device direction
    //modificate vertex buffers with cuda
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t size = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &size, this->cudaResourceBufParticles);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= this->numParticles; k <<= 1) {
        /* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
                bitonic_sort_kernel<<<blocks, threads>>>(this->d_particle, j, k);
        }
    }
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
}

void SystemSPH::ClearGridIndices(){

    //get device direction
    //modificate vertex buffers with cuda
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t size = sizeof(intvec2)*this->numIndices;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &size, this->cudaResourceBufIndices);

    dim3 blocks(this->numIndices%1024==0
         ? int(this->numIndices/1024) : int(this->numIndices/1024)+1 ,1 ,1);    /* Number of blocks   */
    dim3 threads(1024,1,1);  /* Number of threads  */
    clearGridIndices_kernel<<<blocks, threads>>>(this->d_inidices, this->numParticles);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);

}

void SystemSPH::BuiltGridIncices(){
    //get device direction
    //Indices
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t sizeIndices = sizeof(intvec2)*this->numIndices;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &sizeIndices, this->cudaResourceBufIndices);
    //Paricles
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t sizeParticle = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &sizeParticle, this->cudaResourceBufParticles);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */
    builtGridIncices_kernel<<<blocks, threads>>>(this->d_inidices, this->d_particle, this->numParticles);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
}

void SystemSPH::CreateGridIndices(){
    this->ClearGridIndices();
    this->BuiltGridIncices();
}

void SystemSPH::PosParticleCell(){
    //get device direction
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t sizeParticle = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &sizeParticle, this->cudaResourceBufParticles);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */
    posParticleCell_kernel<<<blocks, threads>>>(this->d_particle, this->numParticles,
                                                this->xMeshDim, this->yMeshDim, this->zMeshDim);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
}

void SystemSPH::CalcOperations(){
    //get device direction
    //Indices
    hipGraphicsMapResources(1, &this->cudaResourceBufIndices, 0);
    size_t sizeIndices = sizeof(intvec2)*this->numIndices;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_inidices, &sizeIndices, this->cudaResourceBufIndices);
    //Paricles
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t sizeParticle = sizeof(particle)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &sizeParticle, this->cudaResourceBufParticles);
    //calculation
    hipGraphicsMapResources(1, &this->cudaResourceBufCalc, 0);
    size_t sizeCalc = sizeof(calcStruct)*this->numParticles;
    hipGraphicsResourceGetMappedPointer((void **)&this->d_calc, &sizeCalc, this->cudaResourceBufCalc);

    dim3 blocks(this->numBlocks,1,1);    /* Number of blocks   */
    dim3 threads(this->numThreads,1,1);  /* Number of threads  */
    // calculate operations
    calcul_kernel<<<blocks, threads>>>(this->d_particle, this->d_inidices, this->d_calc, this->numParticles,
                                       this->xMeshDim, this->yMeshDim, this->zMeshDim);
    // integrate (Euler Exlicit)
    integrate_kernel<<<blocks, threads>>>(this->d_particle, this->d_calc, this->numParticles, this->seed);

    hipGraphicsUnmapResources(1, &this->cudaResourceBufIndices, 0);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufCalc, 0);
}

void SystemSPH::Calculate(){
    //ind cell pos indes
    this->PosParticleCell();
    //sort particles
    this->SortParticles();
    //create indices
    this->CreateGridIndices();
    //calculation
    this->CalcOperations();

}

void SystemSPH::SeedUpdate(int i){
    this->seed = i;
}

void SystemSPH::Save(const std::string& nameFile){
    //Paricles
    hipGraphicsMapResources(1, &this->cudaResourceBufParticles, 0);
    size_t size = this->numParticles*sizeof(particle);
    hipGraphicsResourceGetMappedPointer((void **)&this->d_particle, &size, this->cudaResourceBufParticles);
    std::cout << hipMemcpy(this->h_particle, this->d_particle, this->numParticles*sizeof(particle), hipMemcpyDeviceToHost);
    hipGraphicsUnmapResources(1, &this->cudaResourceBufParticles, 0);

    std::ofstream ofs (nameFile, std::ofstream::out);
    //heater
    ofs << "Step (or seed) of simulations" << "," << this->seed << "," << "Number of particles" << "," << this->numParticles << std::endl;
    ofs << "PaticleIndice" << "," << "PaticleCell" << "," << "Position_x" << "," << "Position_y" << ","<< "Position_z" << ",";
    ofs << "Velocity_vx" << "," << "Velocity_vy" << ","<< "Velocity_vz" << std::endl;
    //body
    for(int i=0; i<this->numParticles; i++){
        ofs << this->h_particle[i].id << "," << this->h_particle[i].cellIdx << ",";
        ofs << this->h_particle[i].x << "," << this->h_particle[i].y << "," << this->h_particle[i].z << ",";
        ofs << this->h_particle[i].vx << "," << this->h_particle[i].vy << "," << this->h_particle[i].vz << std::endl;
    }
    ofs.close();

    std::cout << "The current data has been saved in: " << nameFile << std::endl;
}




//////////////////////// Plot /////////////////////////////////////////////////
void SystemSPH::DrawParticles(){
    //openGl
    glBindVertexArray(this->VAO);
    //glDrawArrays(GL_POINTS, 0, this->numParticles);
    glDrawElements(GL_POINTS, this->numParticles, GL_UNSIGNED_SHORT, 0);
    glBindVertexArray(0);
}
void SystemSPH::BackGround(float r, float g, float b, float a){
    glClearColor(r, g, b, a);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
}
void SystemSPH::DrawBoundary(){
    //openGl
    glBindVertexArray(this->VAOboundary);
    //glDrawArrays(GL_POINTS, 0, 8);

    glDrawElements(GL_LINE_LOOP, 16, GL_UNSIGNED_SHORT,  0);
    glBindVertexArray(0);
}
void SystemSPH::PolygonMode(){
    //this call will result in wireframe polygons.
    glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
}

void SystemSPH::CreateBounderiesIndices(){
    // boundaries are drawed using line loop, so:
    this->boundaryIndices[0] = 0;
    this->boundaryIndices[1] = 3;
    this->boundaryIndices[2] = 2;
    this->boundaryIndices[3] = 1;
    this->boundaryIndices[4] = 0;
    this->boundaryIndices[5] = 4;
    this->boundaryIndices[6] = 7;
    this->boundaryIndices[7] = 6;
    this->boundaryIndices[8] = 5;
    this->boundaryIndices[9] = 4;
    this->boundaryIndices[10] = 7;
    this->boundaryIndices[11] = 3;
    this->boundaryIndices[12] = 2;
    this->boundaryIndices[13] = 6;
    this->boundaryIndices[14] = 5;
    this->boundaryIndices[15] = 1;
}

void SystemSPH::CreateBounderiesPoints(){
    //point 0
    this->boundaryPoints[0].x = -1.0;
    this->boundaryPoints[0].y = -1.0;
    this->boundaryPoints[0].z = -1.0;
    //point 1
    this->boundaryPoints[1].x = 1.0;
    this->boundaryPoints[1].y = -1.0;
    this->boundaryPoints[1].z = -1.0;
    //point 2
    this->boundaryPoints[2].x = 1.0;
    this->boundaryPoints[2].y = 1.0;
    this->boundaryPoints[2].z = -1.0;
    //point 3
    this->boundaryPoints[3].x = -1.0;
    this->boundaryPoints[3].y = 1.0;
    this->boundaryPoints[3].z = -1.0;
    //point 4
    this->boundaryPoints[4].x = -1.0;
    this->boundaryPoints[4].y = -1.0;
    this->boundaryPoints[4].z = 1.0;
    //point 5
    this->boundaryPoints[5].x = 1.0;
    this->boundaryPoints[5].y = -1.0;
    this->boundaryPoints[5].z = 1.0;
    //point 6
    this->boundaryPoints[6].x = 1.0;
    this->boundaryPoints[6].y = 1.0;
    this->boundaryPoints[6].z = 1.0;
    //point 7
    this->boundaryPoints[7].x = -1.0;
    this->boundaryPoints[7].y = 1.0;
    this->boundaryPoints[7].z = 1.0;
}
